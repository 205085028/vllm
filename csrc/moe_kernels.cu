#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

//
#include <iostream>

#include "dispatch_utils.h"

#include <c10/util/BFloat16.h>
#include <c10/cuda/CUDAStream.h>

// #include "cutlass/platform/platform.h"
// #include "cutlass/bfloat16.h"
// #include "cutlass/complex.h"
// #include "cutlass/gemm/kernel/gemm_grouped.h"
// #include "cutlass/gemm/kernel/default_gemm_grouped.h"
// #include "cutlass/gemm/device/gemm_grouped.h"

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/group_array_problem_shape.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/device_memory.h"

using namespace cute;

namespace vllm {

#define CUDA_CALL(code)					                    \
  do {                                                      \
    hipError_t status = code;                              \
    std::string err = hipGetErrorString(status);           \
    TORCH_CHECK(status == hipSuccess, err);		        \
  } while (0)

#define GROUPED_GEMM_STRINGIFY_HELPER(x) #x
#define GROUPED_GEMM_STRINGIFY(x) \
  GROUPED_GEMM_STRINGIFY_HELPER(x)

using ProblemShape = cutlass::gemm::GroupProblemShape<Shape<int,int,int>>;  // <M,N,K> per group
using ElementA = cutlass::bfloat16_t;                                       // Element type for A matrix operand
using ElementB = cutlass::bfloat16_t;                                       // Element type for B matrix operand
using ElementC = float;                                                     // Element type for C and D matrix operands

// A matrix configuration
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         LayoutB     = cutlass::layout::RowMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         LayoutC     = cutlass::layout::RowMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TileShape           = Shape<_128,_128,_64>;                           // Threadblock-level tile size
using ClusterShape        = Shape<_1,_1,_1>;                                // Shape of the threadblocks in a cluster
using StageCountType = cutlass::gemm::collective::StageCountAuto;           // Stage count maximized based on the tile size
using KernelSchedule = cutlass::gemm::KernelGroupTmaWarpSpecializedCooperative; // Kernel to launch
using EpilogueSchedule = cutlass::epilogue::NoSmemWarpSpecializedGroup;                     // Epilogue to launch

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShape, ClusterShape,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementAccumulator,
    ElementC, LayoutC, AlignmentC,
    ElementC, LayoutC, AlignmentC,
    EpilogueSchedule
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    KernelSchedule
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    ProblemShape,
    CollectiveMainloop,
    CollectiveEpilogue
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

std::vector<typename ProblemShape::UnderlyingProblemShape> MakeProblemSizes(torch::Tensor b, torch::Tensor batch_sizes) {
  const size_t num_experts = batch_sizes.size(0);
  const size_t k = b.size(1), n = b.size(2);
  std::vector<typename ProblemShape::UnderlyingProblemShape> problem_sizes(num_experts);
  for (int i = 0; i < num_experts; ++i) {
    int64_t batch_size = batch_sizes.data_ptr<int64_t>()[i];
    problem_sizes[i] = {batch_size, n, k};
  }
  return problem_sizes;
}

template <typename T>
torch::Tensor CopyToDevice(const std::vector<T> &x, const torch::Device &device) {
  size_t bytes = x.size() * sizeof(T);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(device);
  torch::Tensor out = torch::empty(bytes, options);

  CUDA_CALL(hipMemcpyAsync(out.data_ptr(),
			    x.data(), bytes,
			    hipMemcpyHostToDevice,
			    c10::cuda::getCurrentCUDAStream()));
  return out;
}

template <typename Gemm>
struct ProblemData {
  std::vector<typename ProblemShape::UnderlyingProblemShape> problem_sizes_host;
  cutlass::DeviceAllocation<typename ProblemShape::UnderlyingProblemShape> problem_sizes;
  cutlass::DeviceAllocation<typename Gemm::ElementA *> ptr_A;
  cutlass::DeviceAllocation<typename Gemm::ElementB *> ptr_B;
  cutlass::DeviceAllocation<typename Gemm::ElementC *> ptr_C;
  cutlass::DeviceAllocation<typename Gemm::GemmKernel::StrideA> stride_A;
  cutlass::DeviceAllocation<typename Gemm::GemmKernel::StrideB> stride_B;
  cutlass::DeviceAllocation<typename Gemm::GemmKernel::StrideC> stride_C;
};

template <typename T>
void CopyDataToDevice(const std::vector<T> &src, cutlass::DeviceAllocation<T> &target) {
  target.resize(src.size());
  target.copy_from_host(target.data());
}

template <typename Gemm>
typename Gemm::Arguments MakeArguments(ProblemData<Gemm>& problem_data,
               torch::Tensor a,
				       torch::Tensor b,
				       torch::Tensor c,
				       torch::Tensor batch_sizes) {
  problem_data.problem_sizes_host = MakeProblemSizes(b, batch_sizes);

  // Calculate the number of threadblocks to use and validate the result.
  int64_t num_experts = problem_data.problem_sizes_host.size();

  std::cout << "num_experts = " << num_experts << std::endl;

  // Create the host arrays of leading dimension data and pointer data.
  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;

  std::vector<int64_t>  offsets_a(num_experts);
  std::vector<int64_t> offsets_b(num_experts);
  std::vector<int64_t> offsets_c(num_experts);
  std::vector<StrideA> stride_a_host;
  std::vector<StrideB> stride_b_host;
  std::vector<StrideC> stride_c_host;
  int64_t elements_a = 0, elements_b = 0, elements_c = 0;

  using ElementA = typename Gemm::ElementA;
  using ElementB = typename Gemm::ElementB;
  using ElementC = typename Gemm::ElementC;
  std::vector<ElementA *> ptr_a_host(num_experts);
  std::vector<ElementB *> ptr_b_host(num_experts);
  std::vector<ElementC *> ptr_c_host(num_experts);

  for (int i = 0; i < num_experts; ++i) {
    auto problem = problem_data.problem_sizes_host[i];
    auto M = get<0>(problem);
    auto N = get<1>(problem);
    auto K = get<2>(problem);

    std::cout << "i = " << i << std::endl;
    std::cout << "M = " << M << std::endl;
    std::cout << "N = " << N << std::endl;
    std::cout << "K = " << K << std::endl;

    auto sa = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, Int<1>{}));
    std::cout << "sa[0]" << get<0>(sa) << std::endl;
    std::cout << "sa[1]" << get<1>(sa) << std::endl;
    std::cout << "sa[2]" << get<2>(sa) << std::endl;
    stride_a_host.push_back(sa);
    stride_b_host.push_back(cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, Int<1>{})));
    stride_c_host.push_back(cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, Int<1>{})));

    offsets_a[i] = elements_a;
    offsets_b[i] = elements_b;
    offsets_c[i] = elements_c;

    ptr_a_host[i] = (ElementA*)a.data_ptr() + offsets_a[i];
    ptr_b_host[i] = (ElementB*)b.data_ptr() + offsets_b[i];
    ptr_c_host[i] = (ElementC*)c.data_ptr() + offsets_c[i];

    elements_a += M * K;
    elements_b += K * N;
    elements_c += M * N;
  }

  // Copy the problem sizes, pointers and leading dimension data to the device.
  CopyDataToDevice(problem_data.problem_sizes_host, problem_data.problem_sizes);

  CopyDataToDevice(ptr_a_host, problem_data.ptr_A);
  CopyDataToDevice(ptr_b_host, problem_data.ptr_B);
  CopyDataToDevice(ptr_c_host, problem_data.ptr_C);

  CopyDataToDevice(stride_a_host, problem_data.stride_A);
  CopyDataToDevice(stride_b_host, problem_data.stride_B);
  CopyDataToDevice(stride_c_host, problem_data.stride_C);

  cutlass::KernelHardwareInfo hw_info;
  hw_info.device_id = b.device().index();
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGrouped,
    {static_cast<int>(num_experts), problem_data.problem_sizes.get(), problem_data.problem_sizes_host.data()},
    {problem_data.ptr_A.get(), problem_data.stride_A.get(),
     problem_data.ptr_B.get(), problem_data.stride_B.get()},
    {{/*alpha=*/1.0f, /*beta=*/0.0f},
     problem_data.ptr_C.get(), problem_data.stride_C.get(),
     problem_data.ptr_C.get(), problem_data.stride_C.get()},
    hw_info
  };

  return arguments;
}

torch::Tensor CutlassGroupedGemm(torch::Tensor a,
				 torch::Tensor b,
				 torch::Tensor c,
				 torch::Tensor batch_sizes) {
  Gemm gemm;
  ProblemData<Gemm> problem_data;

  auto arguments = MakeArguments<Gemm>(problem_data, a, b, c, batch_sizes);
  int64_t workspace_size = gemm.get_workspace_size(arguments);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(a.device());
  torch::Tensor workspace = torch::empty(workspace_size, options);

  // Check if the problem size is supported or not
  auto status = gemm.can_implement(arguments);
  TORCH_CHECK(status == cutlass::Status::kSuccess, cutlass::cutlassGetStatusString(status));

  // Initialize the kernel.
  if(gemm.initialize(arguments, workspace.data_ptr()) != cutlass::Status::kSuccess) {
    TORCH_CHECK(false, "Failed to initialize CUTLASS Grouped GEMM");
  }

  // Execute the kernel in the current stream.
  if(gemm.run(c10::cuda::getCurrentCUDAStream()) != cutlass::Status::kSuccess) {
    TORCH_CHECK(false, "Failed to run CUTLASS Grouped GEMM");
  }
  return c;
}

}

void fused_moe(
    torch::Tensor A,
    torch::Tensor B,
    torch::Tensor C,
    torch::Tensor topk_weights,
    torch::Tensor topk_ids //,
    // torch::Tensor sorted_token_ids,
    // torch::Tensor expert_ids,
    // torch::Tensor num_tokens_post_padded,
    // bool MUL_ROUTED_WEIGHT,
    // int top_k,
    // int parallelism
    ) {
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    
    vllm::CutlassGroupedGemm(A, B, C, topk_weights);
}