#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "dispatch_utils.h"

namespace vllm {
template <typename T>
__global__ void dequant_add_residual_kernel(const int32_t *__restrict__ input,
                                            const T *__restrict__ residual,
                                            T *__restrict__ output,
                                            const float scale, int m, int n) {
  const int tid = threadIdx.x;
  for (int i = tid; i < n; i += blockDim.x) {
    output[blockIdx.x * n + i] =
        (T)((((float)input[blockIdx.x * n + i]) * scale) +
            (float)residual[blockIdx.x * n + i]);
  }
}
} // namespace vllm

void invoke_dequant_add_residual(
    torch::Tensor &out,      // [num_tokens, hidden_size]
    torch::Tensor &input,    // [num_tokens, hidden_size]
    torch::Tensor &residual, // [num_tokens, hidden_size]
    float scale) {
  int m = input.size(0);
  int n = input.size(1);
  dim3 grid(m);
  dim3 block(min(n, 1024));

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      residual.scalar_type(), "dequant_add_residual_kernel", [&] {
        vllm::dequant_add_residual_kernel<scalar_t><<<grid, block, 0, stream>>>(
            input.data_ptr<int32_t>(), residual.data_ptr<scalar_t>(),
            out.data_ptr<scalar_t>(), scale, m, n);
      });
}
